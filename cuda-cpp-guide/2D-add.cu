#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 16

// Kernel definition
__global__ void MatAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    int idx = i * N + j;
    C[idx] = A[idx] + B[idx];
}

int main()
{
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);
    
    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int idx = i * N + j;
            h_A[idx] = i + j;
            h_B[idx] = i * j + 1;
        }
    }
    
    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    // Copy host arrays to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // Kernel invocation with one block of N * N * 1 threads
    int numBlocks = 1;
    dim3 threadsPerBlock(N, N);
    MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
    
    // Wait for GPU to finish
    hipDeviceSynchronize();
    
    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    // Verify result
    printf("Matrix Addition Results (showing first 5x5 elements):\n");
    bool correct = true;
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            int idx = i * N + j;
            float expected = h_A[idx] + h_B[idx];
            printf("C[%d][%d] = %.2f (expected: %.2f) ", i, j, h_C[idx], expected);
            if (h_C[idx] != expected) {
                correct = false;
            }
        }
        printf("\n");
    }
    
    // Check all results
    for (int i = 0; i < N * N; i++) {
        float expected = h_A[i] + h_B[i];
        if (h_C[i] != expected) {
            correct = false;
            break;
        }
    }
    
    printf("\nMatrix computation result: %s\n", correct ? "CORRECT" : "INCORRECT");
    
    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;
}