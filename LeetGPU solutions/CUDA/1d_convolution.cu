#include <hip/hip_runtime.h>

__global__ void convolution_1d_kernel(const float* input, const float* kernel, float* output,
                                      int input_size, int kernel_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int output_size = input_size - kernel_size + 1;

    if (idx < output_size) {
        float sum = 0.0f;
        for (int k = 0; k < kernel_size; ++k) {
            sum += input[idx + k] * kernel[k];
        }
        output[idx] = sum;
    }
}

// input, kernel, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, const float* kernel, float* output, int input_size, int kernel_size) {
    int output_size = input_size - kernel_size + 1;
    int threadsPerBlock = 256;
    int blocksPerGrid = (output_size + threadsPerBlock - 1) / threadsPerBlock;

    convolution_1d_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, kernel, output, input_size, kernel_size);
    hipDeviceSynchronize();
}